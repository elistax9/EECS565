#include <hip/hip_runtime.h>

#include <cctype>

#include <inttypes.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <iostream>
#include <algorithm>

unsigned long get_elapsed(struct timespec *start, struct timespec *end)
{
    uint64_t dur;
    dur = ((uint64_t)end->tv_sec * 1000000000 + end->tv_nsec) - 
        ((uint64_t)start->tv_sec * 1000000000 + start->tv_nsec);
    return (unsigned long)dur;
}

std::string processCipher( std::string input, std::string key, bool encode )
{
    std::string output = input;

    for(int i=0; i<input.length(); i++)
    {
        int keyValue = (int)key[i%key.length()] - 97;
        int textValue = (int)input[i] - 97;

        if(encode)
            output[i] = (char)( ( ( textValue + keyValue ) % 26 ) + 97 );
        else
            output[i] = (char)( ( ( textValue + ( 26 - keyValue ) ) % 26 ) + 97 );
    }

    return output;
}

void runCipher(bool encode)
{
    std::string input;
    std::string key;

    if(encode)
        std::cout << "Please input the text you wish to encode:" << std::endl;
    else
        std::cout << "Please input the text you wish to decode:" << std::endl;

    std::getline( std::cin, input );

    input.erase( remove_if(input.begin(), input.end(), isspace), input.end() );

    std::string output = input;

    std::cout << "Please input the key:" << std::endl;
    std::getline( std::cin, key );
    std::transform(key.begin(), key.end(), key.begin(), ::tolower);

    std::cout << "Your cipher text is:" << std::endl;

    output = processCipher(input, key, encode);

    std::cout << output << std::endl;
}

int main(int argc, char **argv)
{
    std::string choice = "encode";
    bool loop = true;

    std::cout<<"Would you like to encode or decode?"<<std::endl;
    std::getline( std::cin, choice );

    while(loop)
    {
        if( choice == "encode")
        {
            runCipher(true);
            loop = false;
        }
        else if( choice == "decode")
        {
            runCipher(false);
            loop = false;
        }
        else
        {
            std::cout << "That is not a valid input, please input either 'encode' or 'decode'" << std::endl;
            std::getline( std::cin, choice );
        }
    }

    return 0;
}

