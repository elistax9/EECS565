#include <algorithm>
#include <cctype>
#include <hip/hip_runtime.h>
#include <inttypes.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

/**************************************************************************
 * Calculate time elapsed
 **************************************************************************/
unsigned long get_elapsed(struct timespec *start, struct timespec *end)
{
    uint64_t dur;
    dur = ((uint64_t)end->tv_sec * 1000000000 + end->tv_nsec) - 
        ((uint64_t)start->tv_sec * 1000000000 + start->tv_nsec);
    return (unsigned long)dur;
}//end get_elapsed

/**************************************************************************
 * Encode/Decode Input
 **************************************************************************/
std::string processCipher( std::string input, std::string key, bool encode )
{
    struct timespec tstart, tend;

    clock_gettime(CLOCK_REALTIME, &tstart);

    //take out spaces and change all letters to lowercase
    input.erase( remove_if(input.begin(), input.end(), isspace), input.end() );
    std::transform(input.begin(), input.end(), input.begin(), ::tolower);
    std::transform(key.begin(), key.end(), key.begin(), ::tolower);

    std::string output = input;

    for(int i=0; i<input.length(); i++)
    {
        int keyValue = (int)key[i%key.length()] - 97;
        int textValue = (int)input[i] - 97;

        if(encode)
            output[i] = (char)( ( ( textValue + keyValue ) % 26 ) + 97 );
        else
            output[i] = (char)( ( ( textValue + ( 26 - keyValue ) ) % 26 ) + 97 );//I add so I don't have to deal with absolute values
    }

    clock_gettime(CLOCK_REALTIME, &tend);

    printf("Cipher Processing: %ld usec\n", get_elapsed(&tstart, &tend)/1000);

    return output;
}//end processCipher

/**************************************************************************
 * Run cipher with manual input and key values
 **************************************************************************/
void runCipher(bool encode)
{
    std::string input, output;
    std::string key;

    if(encode)
        std::cout << "Please input the text you wish to encode:" << std::endl;
    else
        std::cout << "Please input the text you wish to decode:" << std::endl;

    std::getline( std::cin, input );

    std::cout << "Please input the key:" << std::endl;
    std::getline( std::cin, key );

    std::cout << "Your cipher text is:" << std::endl;

    output = processCipher(input, key, encode);

    std::cout << output << std::endl;
}//end runCipher

int main(int argc, char **argv)
{
    std::string choice = "encode";
    bool loop = true;

    std::cout<<"Would you like to encode or decode?"<<std::endl;
    std::getline( std::cin, choice );

    while(loop)
    {
        if( choice == "encode")
        {
            runCipher(true);
            loop = false;
        }
        else if( choice == "decode")
        {
            runCipher(false);
            loop = false;
        }
        else
        {
            std::cout << "That is not a valid input, please input either 'encode' or 'decode'" << std::endl;
            std::getline( std::cin, choice );
        }
    }

    return 0;
}//end main